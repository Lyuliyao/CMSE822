
#include <hip/hip_runtime.h>
#include<iostream>
using std::cout; using std::endl;
#include<math.h>
#include<omp.h>
#include <fstream>
#include<chrono>
using namespace std;

#define PI 3.14159265
#define left_b 0
#define right_b 0

float fcn_source(float x){
	return pow(PI,2)*sin(PI*x);
}


float f_l(float x, int n, int N){
	return (x+1)/float(N)/4*fcn_source((x+1)/2/float(N)+(float(n)-1.0)/float(N));
}
float f_r(float x, int n, int N){
	return (1-x)/float(N)/4*fcn_source((x+1)/2/float(N)+float(n)/float(N));
}

__global__ void gauss_seidel(float* dev_mass_matrix, float* dev_left_vector, float* dev_u, float* dev_u1,int width) {
  //calculate the row and column for this element of the matrix
  int row = threadIdx.x + (blockDim.x * blockIdx.x);
  if ((row == 0) ){
      dev_u1[row] = left_b;
  }
  else if ((row == width +1) ){
      dev_u1[row] = right_b;
  }
  else if ((row <= width)) {
    dev_u1[row] = (dev_left_vector[row] - dev_mass_matrix[row*(width+2)+row-1]*dev_u[row-1] - dev_mass_matrix[row*(width+2)+row+1] *dev_u[row+1])/dev_mass_matrix[row*(width+2)+row];
  }
}


int main(int argc, char **argv){
    int N = atoi(argv[1]);
    int thread_cnt = atoi(argv[2]);
	float mass_matrix[(N+2)*(N+2)];
	float *dev_mass_matrix,*dev_left_vecotr,*dev_u,*dev_u1;
	float left_vector[N+2];
	float u[N+2];
    int block_cnt;
  std::chrono::time_point<std::chrono::steady_clock> start, stop;
  using time_span = std::chrono::milliseconds;
	for(int i = 1; i <= N; i++){
		mass_matrix[i*(N+2)+i] = 2*float(N);
		mass_matrix[i*(N+2)+i-1] = -float(N);
		mass_matrix[i*(N+2)+i+1] = -float(N);
		left_vector[i]= f_l(-1/sqrt(3),i-1,N) + f_l(1/sqrt(3),i-1,N);
		left_vector[i]= left_vector[i] + f_r(-1/sqrt(3),i-1,N) + f_r(1/sqrt(3),i-1,N);
	}

	u[0]= left_b;
	u[N] = right_b;
	int bytes1 = (N+2)*(N+2) * sizeof(float);
    int bytes2 = (N+2) * sizeof(float);
    hipMalloc((void **) &dev_mass_matrix, bytes1);
      
    hipMalloc((void **) &dev_left_vecotr, bytes2);
    hipMalloc((void **) &dev_u, bytes2);
    hipMalloc((void **) &dev_u1, bytes2);
    hipMemcpy(dev_mass_matrix, mass_matrix, bytes1, hipMemcpyHostToDevice);
    hipMemcpy(dev_left_vecotr, left_vector, bytes2, hipMemcpyHostToDevice);
    hipMemcpy(dev_u, u, bytes2, hipMemcpyHostToDevice);
    block_cnt = (N+2)/thread_cnt + ((N+2) % thread_cnt > 0);
  start = std::chrono::steady_clock::now();
	for (int t=0; t<100000; t++){
	    gauss_seidel<<<block_cnt, thread_cnt>>>(dev_mass_matrix, dev_left_vecotr, dev_u, dev_u1, N);
	    dev_u = dev_u1;
	}
   stop = std::chrono::steady_clock::now(); 
    auto elapsed_gpu_1 = std::chrono::duration_cast<time_span>(stop - start).count();
	cout << elapsed_gpu_1 << endl;
	ofstream myfile;
	hipMemcpy(u, dev_u, bytes2, hipMemcpyDeviceToHost);
  myfile.open ("result.txt");
	for (int i = 0; i<N+1;i++){
		myfile << u[i] << " ";
	}
	myfile << u[N+1];
  myfile.close();
}
